#include "hip/hip_runtime.h"
#include <torch/extension.h>

template <typename scalar_t>
__global__ void rational_fwd_cuda_kernel_1dgroup(
    const scalar_t* __restrict__ x, 
    const scalar_t* __restrict__ a,
    const scalar_t* __restrict__ b, 
    scalar_t* __restrict__ result, 
    int B, int L, int D, int group, 
    int x_size, int D_per_group, int num_weights) {

    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= x_size) return;  // Prevent out-of-bounds memory access

    int w_index = idx / (B * L * D);
    int local_idx = idx % (B * L * D);
    // Calculate the index within the dimension D
    int d_index = local_idx % D;
    // Calculate the group index based on the position within dimension D
    int g_index = floor(d_index / D_per_group);

    
    

    // Calculate specific indices for a and b based on group
    int a_idx = w_index * 6;
    int b_idx = w_index * 4;
    // Load coefficients into registers
    scalar_t s_a[6], s_b[4];
    #pragma unroll
    for (int i = 0; i < 6; ++i) {
        s_a[i] = a[a_idx + i];
    }
    #pragma unroll
    for (int i = 0; i < 4; ++i) {
        s_b[i] = abs(b[b_idx + i]);  // Store absolute values directly if needed
    }

    // Obtain the input value from the tensor
    scalar_t xp1 = x[idx];
    scalar_t abs_xp1 = abs(xp1);

    // Compute the polynomial for P using Horner's method
    scalar_t P = s_a[5];
    #pragma unroll
    for (int i = 4; i >= 0; --i) {
        P = fmaf(P, xp1, s_a[i]);
    }
    
    // Compute the polynomial for Q using Horner's method
    scalar_t Q = s_b[3];
    #pragma unroll
    for (int i = 2; i >= 0; --i) {
        Q = fmaf(Q, abs_xp1, s_b[i]);
    }
    Q = fmaf(Q, abs_xp1, 1.0);

    // Write the result of P / Q
    result[idx] = P / Q;
}


torch::Tensor rational_fwd_cuda_1dgroup(
    torch::Tensor x, 
    torch::Tensor n, 
    torch::Tensor d,
    int group
    ){
    auto result = at::empty_like(x);
    const int x_size = x.numel();
    int num_weights = x.size(0);
    int B = x.size(1);
    int L = x.size(2);
    int D = x.size(3);

    int threads_per_block = 256;  // Adjust as needed based on device capabilities
    int num_blocks = (x_size + threads_per_block - 1) / threads_per_block;

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(x.scalar_type(), "rational_fwd_cuda_1dgroup", ([&] {
    rational_fwd_cuda_kernel_1dgroup<scalar_t>
        <<<num_blocks, threads_per_block>>>(
            x.data_ptr<scalar_t>(),
            n.data_ptr<scalar_t>(),
            d.data_ptr<scalar_t>(),
            result.data_ptr<scalar_t>(),
            B, L, D, group, x_size, D / group, num_weights);
        }));

    return result;
}

//P(X) = a_0 + a_1*X + a_2*X^2 ...
//Q(X) = 1 + |b_0||X| + |b_1||X|^2 + |b_2||X|^3
//R(X) = a_1 + 2*a_2*X + 3*a_3*X ...
//S(X) = sign(X) * ( |b_0| + 2|b_1||X| + 3|b_2||X|^2 ...)
//dF/dx = (-P(X)/Q(X)^2)*S(X) + R(X)/Q(X)
//dF/da_i = x^i/Q(X), i \in {0,5}
//dF/db_i = (-P(X)/Q(X)^2) * sign(b_i) * |X^{i+1}| , i \in {0,4}



template <typename scalar_t>
__global__ void rational_bwd_cuda_kernel_1dgroup(
    const scalar_t* __restrict__ grad_output,
    const scalar_t* __restrict__ x,
    const scalar_t* __restrict__ a,
    const scalar_t* __restrict__ b,
    scalar_t* __restrict__ d_x,
    float* __restrict__ d_a,
    float* __restrict__ d_b,
    int B, int L, int D, int group, 
    int x_size, 
    const int n_size, 
    const int d_size,
    int D_per_group, 
    int numerator, 
    int denominator, 
    int num_weights) {
    
    // Shared memory for accumulation
    // group < 32, so we can use 192 and 128 shared memory
    
    //hardcoded for OneConvfor now -> biggest layer has 1 in 6 out 9 kernel weights
    // __shared__ float sda[32*6];
    // __shared__ float sdb[32*4];
    extern __shared__ float shared_mem[];
    float* sda = shared_mem;
    float* sdb = sda + (num_weights * 6);

    // initialize shared memory to zero
    if ( threadIdx.x == 0) {
        #pragma unroll
        for (int i = 0; i < num_weights * 6; ++i) {
            sda[i] = 0;
        }
        #pragma unroll
        for (int i = 0; i < num_weights * 4; ++i) {
            sdb[i] = 0;
        }
    }


    __syncthreads();

    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= x_size) return;  // Prevent out-of-bounds memory access

    // Calculate the index within the dimension D
    int w_index = idx / (B * L * D);
    int local_idx = idx % (B * L * D);
    int d_index = local_idx % D;
    // Calculate the group index based on the position within dimension D
    int g_index = floor(d_index / D_per_group);

    //int w_index = (idx / D) % num_weights;

    // Calculate specific indices for a and b based on group
    int a_idx = w_index * 6;
    int b_idx = w_index * 4;

    // Load coefficients into registers
    scalar_t shared_a[6], shared_b_abs[4], shared_b[4];
    #pragma unroll
    for (int i = 0; i < 6; ++i) {
        shared_a[i] = a[a_idx + i];
    }
    #pragma unroll
    for (int i = 0; i < 4; ++i) {
        shared_b_abs[i] = abs(b[b_idx + i]);  // Store absolute values directly if needed
        shared_b[i] = b[b_idx + i];
    }

    scalar_t local_da[6] = {0}; // Local accumulation arrays
    scalar_t local_db[4] = {0};
    
    scalar_t xp = x[idx];
    scalar_t axp = abs(xp);
    // Compute powers of xp
    scalar_t xp_powers[5];
    xp_powers[0] = xp;
    xp_powers[1] = xp * xp_powers[0]; // xp^2
    xp_powers[2] = xp * xp_powers[1]; // xp^3
    xp_powers[3] = xp * xp_powers[2]; // xp^4
    xp_powers[4] = xp * xp_powers[3]; // xp^5

    // Compute powers of axp
    scalar_t axp_powers[4];
    axp_powers[0] = axp;
    axp_powers[1] = axp * axp_powers[0]; // axp^2
    axp_powers[2] = axp * axp_powers[1]; // axp^3
    axp_powers[3] = axp * axp_powers[2]; // axp^4

    // Compute absolute values once

    scalar_t P = shared_a[0] 
    + shared_a[1] * xp_powers[0] 
    + shared_a[2] * xp_powers[1] 
    + shared_a[3] * xp_powers[2] 
    + shared_a[4] * xp_powers[3] 
    + shared_a[5] * xp_powers[4];

    scalar_t Q = scalar_t(1.0)
    + shared_b_abs[0] * axp_powers[0] 
    + shared_b_abs[1] * axp_powers[1] 
    + shared_b_abs[2] * axp_powers[2] 
    + shared_b_abs[3] * axp_powers[3];


    scalar_t R = shared_a[1] 
    + scalar_t(2.0) * shared_a[2] * xp_powers[0] 
    + scalar_t(3.0) * shared_a[3] * xp_powers[1] 
    + scalar_t(4.0) * shared_a[4] * xp_powers[2] 
    + scalar_t(5.0) * shared_a[5] * xp_powers[3];

    scalar_t S = copysign(scalar_t(1.0), xp) * (shared_b_abs[0] 
    + scalar_t(2.0) * shared_b_abs[1] * axp_powers[0] 
    + scalar_t(3.0) * shared_b_abs[2] * axp_powers[1] 
    + scalar_t(4.0) * shared_b_abs[3] * axp_powers[2]);
    

    scalar_t grad_o = grad_output[idx];
    
    scalar_t mpq2 = -P/(Q*Q);

    scalar_t d_i_x = (R / Q + S * mpq2) * grad_o;
    d_x[idx] = d_i_x;

    // Loop for computing d_a contributions
    
    if (numerator == -1) {
        numerator = 2;
    } else {    
        local_da[0] = scalar_t(1.0) / Q * grad_o;
    }

    //local_da[0] = scalar_t(1.0) / Q * grad_o;
    #pragma unroll
    for (int i = 1; i < numerator; ++i) {
        local_da[i] = (xp_powers[i-1] / Q) * grad_o;
    }

    // Loop for computing d_b contributions
    #pragma unroll
    for (int i = 0; i < denominator; ++i) {
        local_db[i] = mpq2 * copysign(scalar_t(1.0), shared_b[i]) * axp_powers[i] * grad_o;
    }

    // Reduce local arrays to shared memory
    #pragma unroll
    for (int i = 0; i < 6; ++i) {
        atomicAdd(&sda[a_idx + i], local_da[i]);
    }
    #pragma unroll
    for (int i = 0; i < 4; ++i) {
        atomicAdd(&sdb[b_idx + i], local_db[i]);
    }

    __syncthreads();

    // Only one thread writes back to global memory
    if (threadIdx.x == 0) {
        #pragma unroll
        for (int i = 0; i < n_size; ++i) {
            atomicAdd(&d_a[i], sda[i]);
        }
        #pragma unroll
        for (int i = 0; i < d_size; ++i) {
            atomicAdd(&d_b[i], sdb[i]);
        }
    }
    
}

std::vector<torch::Tensor> rational_bwd_cuda_1dgroup(torch::Tensor grad_output, torch::Tensor x, torch::Tensor n, torch::Tensor d, int group, int numerator, int denominator) {
    const int x_size = x.numel();
    const int n_size = n.numel();
    const int d_size = d.numel();

    auto d_x = at::empty_like(x);
    auto d_n = at::zeros_like(n).toType(at::kFloat);
    auto d_d = at::zeros_like(d).toType(at::kFloat);

    int num_weights = x.size(0);
    int B = x.size(1);
    int L = x.size(2);
    int D = x.size(3);

    int blockSize = 256;  // You might want to experiment with this value
    int numBlocks = (x_size + blockSize - 1) / blockSize;
    int shared_mem_size = (num_weights * 10) * sizeof(float);

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(x.scalar_type(), "rational_bwd_cuda_1dgroup", ([&] {
    rational_bwd_cuda_kernel_1dgroup<scalar_t>
        <<<numBlocks, blockSize, shared_mem_size>>>(
            grad_output.data_ptr<scalar_t>(),
            x.data_ptr<scalar_t>(),
            n.data_ptr<scalar_t>(),
            d.data_ptr<scalar_t>(),
            d_x.data_ptr<scalar_t>(),
            d_n.data_ptr<float>(),
            d_d.data_ptr<float>(),
            B, L, D, group, x_size, n_size, d_size, D / group, numerator, denominator, num_weights);
    }));

    return {d_x, d_n, d_d};
}

// template <typename scalar_t>
// __global__ void imtocol_kernel(const scalar_t* input, scalar_t* output, int batch_size, int channels, int height, int width, int out_size, int kernel_size) {

//     int idx = blockIdx.x * blockDim.x + threadIdx.x;
//     if (idx >= batch_size * channels * out_size * out_size * kernel_size * kernel_size) return;

//     int wh = width * height;
//     int owh = out_size * out_size;

//     int batch = idx / (channels * owh * kernel_size * kernel_size);
//     int ch_idx = (idx % (channels * owh * kernel_size * kernel_size)) / (owh * kernel_size * kernel_size);
//     int oh_idx = (idx % (owh * kernel_size * kernel_size)) / (out_size * kernel_size * kernel_size);
//     int ow_idx = (idx % (out_size * kernel_size * kernel_size)) / (kernel_size * kernel_size);
//     int kh_idx = (idx % (kernel_size * kernel_size)) / kernel_size;
//     int kw_idx = idx % kernel_size;

//     int h_offset = oh_idx + kh_idx;
//     int w_offset = ow_idx + kw_idx;

//     output[idx] = input[batch * channels * wh + ch_idx * wh + h_offset * width + w_offset];

// }

// torch::Tensor imtocol_cuda(torch::Tensor input, int kernel_size) {

//     int batch_size = input.size(0);
//     int channels = input.size(1);
//     int height = input.size(2); //padded
//     int width = input.size(3); //padded

//     //printf("%d", height);

//     int out_h = height - kernel_size + 1;
//     int out_w = width - kernel_size + 1;

//     auto output = torch::zeros({batch_size, channels, out_h, out_w, kernel_size, kernel_size}, torch::device(input.device()).dtype(input.dtype()));

//     int num_kernels = batch_size * channels * out_h * out_w * kernel_size * kernel_size;
//     int block_size = 256;
//     int num_blocks = (num_kernels + block_size - 1) / block_size;

//     AT_DISPATCH_FLOATING_TYPES_AND_HALF(input.scalar_type(), "imtocol_kernel", ([&] {
//         imtocol_kernel<scalar_t><<<num_blocks, block_size>>>(input.data_ptr<scalar_t>(), output.data_ptr<scalar_t>(), batch_size, channels, height, width, out_h, kernel_size);}));

//     return output;

// }